// Test run kernel in kernel

#include <stdio.h>
#include "cudaheader.cuh"
#include "kernel.cuh"

__device__ int v = -123;



hipError_t runTest1()
{
    double * tempArray;
    size_t num_elements = 1024;
    hipMallocManaged(&tempArray, num_elements*sizeof(double));


    kernel_64reg<<<1,512>>>(tempArray);
    hipError_t ret = hipGetLastError();
    gpuErrchk("test1 512 threads/block launch", ret);
    ret = hipDeviceSynchronize();
    gpuErrchk("test1 512 threads/block sync", ret);

    hipFree(tempArray);
    
    return ret;
}

hipError_t runTest2()
{
    double * tempArray;
    size_t num_elements = 1024;
    hipMallocManaged(&tempArray, num_elements*sizeof(double));


    kernel_64reg<<<1,1024>>>(tempArray);
    hipError_t ret = hipGetLastError();
    gpuErrchk("test2 1024 threads/block launch", ret);
    ret = hipDeviceSynchronize();
    gpuErrchk("test2 1024 threads/block sync", ret);

    hipFree(tempArray);
    
    return ret;
}

int main()
{
    printf("=====================================\n");
    
    if (runTest1() != hipSuccess)
        printf ("xxxFAILED Test 1\n");
    else
        printf ("SUCCEEDED Test 1\n");

    
    if (runTest2() != hipSuccess)
        printf ("xxxFAILED Test 2\n");
    else
        printf ("SUCCEEDED Test 2\n");

}
